#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstdint>
#include <cstdio>
#include <execution>
#include <random>
#include <string>
#include <utility>
#include <vector>

#include "cuda_api.cu"
#include "support.cu"

#define BLOCK_SZ 512U

std::size_t
div_ceil(std::size_t const val, std::size_t divisor)
// https://stackoverflow.com/questions/2745074/fast-ceiling-of-an-integer-division-in-c-c
{
    return val ? 1 + ((val - 1) / divisor) : val;
}

unsigned
is_point_in_polygon(float const pq_x, float const pq_y,
                    float const * const polygon_x,
                    float const * const polygon_y,
                    unsigned long long polygon_vertex_count)
{
    unsigned intersect_count{ 0U };
    for(unsigned long long i{ 0U }, j{ polygon_vertex_count - 1U };
        i < polygon_vertex_count; ++i)
    {
        float const pj_x{ polygon_x[j] };
        float const pj_y{ polygon_y[j] };
        float const pi_x{ polygon_x[i] };
        float const pi_y{ polygon_y[i] };
        // https://wrf.ecse.rpi.edu/Research/Short_Notes/pnpoly.html
        intersect_count
            += ((pi_y > pq_y) != (pj_y > pq_y))
               && (pq_x < (pj_x - pi_x) * (pq_y - pi_y) / (pj_y - pi_y) + pi_x);
        j = i;
    }
    return intersect_count & 0x01U;
}

__global__ void
are_points_in_polygon_kernel(float const * const points_x,
                             float const * const points_y,
                             unsigned long long const point_count,
                             float const * const polygon_x,
                             float const * const polygon_y,
                             unsigned long long polygon_vertex_count,
                             unsigned * const are_points_in_polygon_out)
{
    // uses dynamic shared memory
    // https://developer.nvidia.com/blog/using-shared-memory-cuda-cc/
    extern __shared__ float dynamic_shmem[];
    float * const local_polygon_x = dynamic_shmem;
    float * const local_polygon_y = local_polygon_x + polygon_vertex_count;

    unsigned int const block_idx_x = blockIdx.x;
    unsigned int const block_dim_x = blockDim.x;
    unsigned int const grid_dim_x = gridDim.x;
    unsigned int const thread_count = block_dim_x * grid_dim_x;
    unsigned int const t_idx_x = threadIdx.x;
    for(unsigned int i = t_idx_x; i < polygon_vertex_count; i += block_dim_x)
    {
        local_polygon_x[i] = polygon_x[i];
        local_polygon_y[i] = polygon_y[i];
    }
    __syncthreads();
    unsigned int const begin_idx = t_idx_x + block_idx_x * block_dim_x;
    for(unsigned int i = begin_idx; i < point_count; i += thread_count)
    {
        float const pq_x = points_x[i];
        float const pq_y = points_y[i];
        are_points_in_polygon_out[t_idx_x] = is_point_in_polygon(
            pq_x, pq_y, local_polygon_x, local_polygon_y, polygon_vertex_count);
    }
}

// issue changed interface since cpp stl types were not easily
// transferable using the cuda memory library primitives. Concern that
// c-style code will propigate into cpp application when using cuda.
void
are_points_in_polygon(float const * const points_x_h,
                      float const * const points_y_h,
                      unsigned long long const point_count,
                      float const * const polygon_x_h,
                      float const * const polygon_y_h,
                      unsigned long long polygon_vertex_count,
                      unsigned * const are_points_in_polygon_out_h)
{
    if(polygon_vertex_count < 3U)
    {
        return;
    }
    if(!point_count)
    {
        return;
    }
    unsigned long long const point_count_bytes = point_count * sizeof(float);
    float * points_x_d = (float *)(cuda_malloc(point_count_bytes));
    float * points_y_d = (float *)(cuda_malloc(point_count_bytes));

    unsigned long long const polygon_vertex_count_bytes
        = polygon_vertex_count * sizeof(float);
    float * polygon_x_d = (float *)(cuda_malloc(polygon_vertex_count_bytes));
    float * polygon_y_d = (float *)(cuda_malloc(polygon_vertex_count_bytes));

    unsigned long long const out_bytes = point_count * sizeof(unsigned);
    unsigned * const are_points_in_polygon_out_d
        = (unsigned * const)(cuda_malloc(out_bytes));

    cuda_push(points_x_h, points_x_d, point_count_bytes);
    cuda_push(points_y_h, points_y_d, point_count_bytes);
    cuda_push(polygon_x_h, polygon_x_d, polygon_vertex_count_bytes);
    cuda_push(polygon_y_h, polygon_y_d, polygon_vertex_count_bytes);

    std::size_t dynamic_shared_memory_sz = 2 * point_count_bytes;
    std::size_t const grid_sz{ div_ceil(point_count, BLOCK_SZ) };
    dim3 dim_grid(grid_sz, 1, 1);
    dim3 dim_block(BLOCK_SZ, 1, 1);

    hipDeviceSynchronize();

    are_points_in_polygon_kernel<<<dim_grid, dim_block,
                                   dynamic_shared_memory_sz>>>(
        points_x_d, points_y_d, point_count, polygon_x_d, polygon_y_d,
        polygon_vertex_count, are_points_in_polygon_out_d);

    hipError_t cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess)
    {
        // bad but works for now
        FATAL("Unable to launch kernel");
    }

    cuda_pull(are_points_in_polygon_out_d, are_points_in_polygon_out_h,
              out_bytes);
    cuda_free(points_x_d);
    cuda_free(points_y_d);
    cuda_free(polygon_x_d);
    cuda_free(polygon_y_d);
    cuda_free(are_points_in_polygon_out_d);
}

bool
test_unit_square()
{
    puts("begin test_unit_square");
    unsigned long long point_count = 1000000000;
    puts("begin point allocation");
    unsigned long long const point_count_bytes = point_count * sizeof(float);
    float * points_x = (float *)(malloc(point_count_bytes));
    float * points_y = (float *)(malloc(point_count_bytes));

    unsigned long long const out_bytes = point_count * sizeof(unsigned);
    unsigned * const are_points_in_polygon_out
        = (unsigned * const)(malloc(out_bytes));

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-1.0, 2.0);
    puts("begin point generation");
    for(std::size_t i{ 0U }; i < point_count; ++i)
    {
        points_x[i] = dis(gen);
        points_y[i] = dis(gen);
        are_points_in_polygon_out[i] = 0;
    }
    puts("init unit square");
    float unit_square_x[] = { 0.0, 0.0, 1.0, 1.0 };
    float unit_square_y[] = { 0.0, 1.0, 1.0, 0.0 };
    unsigned long long polygon_vertex_count = 4;

    puts("begin point testing");
    Timer timer;
    startTime(&timer);
    are_points_in_polygon(points_x, points_y, point_count, unit_square_x,
                          unit_square_y, polygon_vertex_count,
                          are_points_in_polygon_out);
    stopTime(&timer);
    printf("%f s\n", elapsedTime(timer));
    puts("begin point verification");
    for(std::size_t i{ 0U }; i < point_count; ++i)
    {
        unsigned const s0{ (0.0 < points_x[i] && points_x[i] < 1.0
                            && 0.0 < points_y[i] && points_y[i] < 1.0) };
        if(s0 != are_points_in_polygon_out[i])
        {
            printf("%lu : (%f,%f) : %u =\\= %u\n", i, points_x[i], points_y[i],
                   s0, are_points_in_polygon_out[i]);
            return false;
        }
    }
    return true;
}

int
main()
{
    // run test trials
    for(int i{ 0 }; i < 32; ++i)
    {
        printf("################# TEST : %d\n", i);
        bool rc = test_unit_square();
        if(rc)
        {
            puts("test_unit_square passed");
        }
        else
        {
            puts("test_unit_square failed");
        }
    }
}