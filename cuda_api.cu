#include "cuda_api.h"

void *
cuda_malloc(std::size_t const bytes)
{
    void * ret{ nullptr };
    hipError_t const rc{ hipMalloc(&ret, bytes) };
    if(rc != hipSuccess)
    {
        return nullptr;
    }
    return ret;
}

void
cuda_free(void * ptr)
{
    hipFree(ptr);
}

void *
cuda_memcpy(void * const dest, void const * const src, std::size_t const bytes,
            hipMemcpyKind const kind)
{
    hipError_t const rc{ hipMemcpy(
        const_cast<void *>(dest), const_cast<void const *>(src), bytes, kind) };
    if(rc != hipSuccess)
    {
        return nullptr;
    }
    return dest;
}

void *
cuda_push(void const * const from_host, void * const to_device,
          std::size_t const bytes)
{
    return cuda_memcpy(to_device, from_host, bytes, hipMemcpyHostToDevice);
}

void *
cuda_pull(void const * const from_device, void * const to_host,
          std::size_t const bytes)
{
    return cuda_memcpy(to_host, from_device, bytes, hipMemcpyDeviceToHost);
}
